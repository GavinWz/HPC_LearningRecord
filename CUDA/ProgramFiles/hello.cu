#include "hip/hip_runtime.h"
#include "error.cuh"
#include<stdio.h>
__global__ void myKernel(){
    printf("Hello World.\n");
}

int main(){
    myKernel<<<2, 2>>>();
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    // hipDeviceReset();
    return 0;
}