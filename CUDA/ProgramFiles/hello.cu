#include "hip/hip_runtime.h"
#include "error.cuh"
#include<stdio.h>
__global__ void myKernel(){
    printf("Hello World.\n");
}

int main(){
    myKernel<<<1, 2>>>();
    CHECK(hipDeviceSynchronize());
    // hipDeviceReset();
    return 0;
}