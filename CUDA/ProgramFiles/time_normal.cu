#include<stdio.h>
#include<stdlib.h>
#include "error.cuh"

void addArrays(float *a, float *b, float *c, int N);

int main(){
    const int N = 100000;
    const int M = sizeof(float) * N;
    float *h_a, *h_b, *h_c;
    float value_a = 1.11, value_b = 2.22;
    int i;

    h_a = (float*)malloc(M);
    h_b = (float*)malloc(M);
    h_c = (float*)malloc(M);

    for(i = 0; i < N; i++)
        h_a[i] = value_a;
    for(i = 0; i < N; i++)
        h_b[i] = value_b;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    hipEventQuery(start);

    addArrays(h_a, h_b, h_c, N);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float time_passed;
    hipEventElapsedTime(&time_passed, start, end);
    printf("Time passed %f ms.\n", time_passed);
    
    hipEventDestroy(start);
    hipEventDestroy(end);

    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}

void addArrays(float *a, float *b, float *c, int N){
    for(int i = 0; i < N; i++){
        c[i] = a[i] + b[i];
    }
}