
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void printId(){
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    printf("I come from block %d - thread %d.\n", blockId, threadId);
}

int main(){
    printId<<<2, 4>>>();
    return 0;
} 