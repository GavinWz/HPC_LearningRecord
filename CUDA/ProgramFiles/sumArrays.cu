
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void sumKernel(double *d_a, double *d_b, double *d_c, int n);

double *h_a, *h_b, *h_c;
const int N = 1000;
const int M = sizeof(double) * N;

int main(){
    h_a = (double*)malloc(M);
    h_b = (double*)malloc(M);
    h_c = (double*)malloc(M);
    
    for(int i = N - 1; i >= 0; i--)
        h_a[i] = i;
    for(int i = 0; i < N; i++)
        h_b[i] = i;

    double *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, M);
    hipMalloc((void**)&d_b, M);
    hipMalloc((void**)&d_c, M);
    hipMemcpy(d_a, h_a, M, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, M, hipMemcpyHostToDevice);

    const int block_size = 64;
    const int grid_size = N / block_size + 1;
    sumKernel<<<grid_size, block_size>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, M, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++)
        printf("%lf + %lf = %lf\n",h_a[i], h_b[i], h_c[i]);

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
__global__ void sumKernel(double *d_a, double *d_b, double *d_c, int n){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    d_c[index] = d_a[index] + d_b[index];
}