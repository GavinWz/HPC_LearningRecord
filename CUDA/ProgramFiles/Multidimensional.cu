
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void multiThreads(){
    const int block = blockIdx.x;
    const int threadx = threadIdx.x;
    const int thready = threadIdx.y;
    printf("I come from block %d and thread(%d, %d).\n", block, threadx, thready);
}

int main(){
    dim3 block_size(2, 4);
    multiThreads<<<1, block_size>>>();
    return 0;
}