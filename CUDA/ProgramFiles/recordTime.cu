#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include "error.cuh"

__global__ void addArrays(float *a, float *b, float *c, int N);

int main(){
    const int N = 100000;
    const int M = sizeof(float) * N;
    float *h_a, *h_b, *h_c;
    float value_a = 1.11, value_b = 2.22;
    int i;

    h_a = (float*)malloc(M);
    h_b = (float*)malloc(M);
    h_c = (float*)malloc(M);

    for(i = 0; i < N; i++)
        h_a[i] = value_a;
    for(i = 0; i < N; i++)
        h_b[i] = value_b;

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, M);
    hipMalloc((void**)&d_b, M);
    hipMalloc((void**)&d_c, M);
    hipMemcpy(d_a, h_a, M, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, M, hipMemcpyHostToDevice);

    int block_size = 64;
    int grid_size = (N + block_size - 1) / block_size;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    hipEventQuery(start);

    addArrays<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, M, hipMemcpyDeviceToHost);
    // for(i = 0; i < N;i++)
    //     printf("%f\n", h_c[i]);

    hipEventRecord(end);
    hipEventSynchronize(end);
    float time_passed;
    hipEventElapsedTime(&time_passed, start, end);
    printf("Time passed %f ms.\n", time_passed);

}

__global__ void addArrays(float *a, float *b, float *c, int N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
        c[index] = a[index] + b[index];
}